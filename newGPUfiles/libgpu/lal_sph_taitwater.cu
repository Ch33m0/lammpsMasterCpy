#include "hip/hip_runtime.h"
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
#else
texture<int4,1> pos_tex;
#endif
#else
#define pos_tex x_
#endif

#define MY_PI (acctyp)3.14159265358979323846

__kernel void k_sph_taitwater(const __global numtyp4 *restrict x_,
                     const __global numtyp4 *restrict coeff,
                     const int lj_types,
                     const __global numtyp *restrict sp_lj_in,
                     const __global int *dev_nbor,
                     const __global int *dev_packed,
                     __global acctyp4 *restrict ans,
                     __global acctyp *restrict engv,
                     const int eflag, const int vflag, const int inum,
                     const int nbor_pitch, const int t_per_atom) {
  int tid, ii, offset;
  
  //CP: needs to be modified to have more atom info for SPH
  
  atom_info(t_per_atom,ii,tid,offset);

//CP: sp_lj and sp_lj_in would not be used for SPH taitwater
  __local numtyp sp_lj[4];
  sp_lj[0]=sp_lj_in[0];
  sp_lj[1]=sp_lj_in[1];
  sp_lj[2]=sp_lj_in[2];
  sp_lj[3]=sp_lj_in[3];

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

//CP: would need a fetch4 for i atom velocity vector as well
    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;


//CP: here would need to access rho for computer Tait EOS force of i atom 

    numtyp factor_lj;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      factor_lj = sp_lj[sbmask(j)];
      j &= NEIGHMASK;

//CP: would need a fetch4 for j atom velocity vector as well
      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      
//CP: below here, this would need to be all different, in style of SPH Taitwater,
// would need to have fj EOS computation, then the artificial viscosity calculations, then different equation for change in force
//CP: also need to add equation to change density and thermal energy of system, then add/modify all methods that store this data back from GPU

      int mtype=itype*lj_types+jtype;
      if (rsq<coeff[mtype].z) {
        numtyp force;
        numtyp r = ucl_sqrt(rsq);
        numtyp arg = MY_PI*r/coeff[mtype].y;
        if (r > (numtyp)0.0) force = factor_lj * coeff[mtype].x *
                       sin(arg) * MY_PI/coeff[mtype].y*ucl_recip(r);
        else force = (numtyp)0.0;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        if (eflag>0) {
          numtyp e=coeff[mtype].x * ((numtyp)1.0+cos(arg));
          energy+=factor_lj*e;
        }
        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

//CP: removed fast version, no point in keeping it
//!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!